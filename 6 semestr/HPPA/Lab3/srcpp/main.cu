
#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <ostream>
#include <sstream>
#include <string>
#include <thread>

#include <chrono>
#include <fstream>

using std::cerr;
using std::cout;
using std::string;

size_t maxThreadsPerBlock = 64; //! must be a power of 2 and max 1024

__global__ void calculate(unsigned int *arrayGPU, unsigned int *resultGPU, size_t arraySize) {
	unsigned long index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index > pow(arraySize / 2, 2))
		return; // if index is out of range

	size_t increment = arraySize / 2;
	increment = index / increment * arraySize;

	resultGPU[index * 4] = arrayGPU[index * 2 + increment + arraySize + 1];
	resultGPU[index * 4 + 1] = arrayGPU[index * 2 + increment + arraySize];
	resultGPU[index * 4 + 2] = arrayGPU[index * 2 + increment + 1];
	resultGPU[index * 4 + 3] = arrayGPU[index * 2 + increment];
}

/**
 * @brief Calculates grid and block size for CUDA kernel
 *
 * @param arraySize
 * @return unsigned*
 */
unsigned int *calcGridBlockSize(size_t arraySize) {
	unsigned int *result = new unsigned int[2];

	if (pow(arraySize / 2, 2) > maxThreadsPerBlock) {
		result[0] = pow(arraySize / 2, 2) / maxThreadsPerBlock;
		result[1] = maxThreadsPerBlock;
	} else {
		result[0] = 1;
		result[1] = pow(arraySize / 2, 2);
	}

	return result;
}

int main(int argc, char **argv) {
	size_t arraySize;
	bool nfs = false;
	if (argc == 1) {
		cout << "Lab3 - CUDA" << std::endl
				 << "Usage: ./run <arraySize> <options>" << std::endl
				 << "Options:" << std::endl
				 << "nfs - no file save, program won't write output of functions into files" << std::endl
				 << "threads <number> - set number of threads per block (default 64)" << std::endl
				 << "Example: ./run 1024 nfs threads 64" << std::endl
				 << "-----------------------------------------------------------------------" << std::endl;
		return 0;
	} else { // checkers
		try {
			arraySize = std::stoul(argv[1]);
		} catch (...) {
			cerr << "Failed to convert '" << argv[1] << "' to size_t" << std::endl;
			return 1;
		}

		for (int i = 2; i < argc; i++) {
			if (argv[i] == string("nfs")) {
				nfs = true;
			} else if (argv[i] == string("threads")) {
				try {
					maxThreadsPerBlock = std::stoul(argv[i + 1]);
					if (maxThreadsPerBlock % 2 != 0)
						throw("Threads per block must be a power of 2");
					else if (maxThreadsPerBlock > 1024)
						throw("Threads per block must be less than 1024");
					else if (maxThreadsPerBlock == 0)
						throw("Threads per block must be greater than 0");
				} catch (string error) {
					cout << error << std::endl;
					return 1;
				} catch (...) {
					cerr << "Failed to convert '" << argv[i + 1] << "' to size_t" << std::endl;
					return 1;
				}
			}
		}
	}
	if ((arraySize == 0) || (arraySize % 4 != 0)) {
		cout << "Array size must be a multiple of 4" << std::endl;
		return 1;
	} else
		cout << "Size is set to " << arraySize * arraySize << std::endl;

	const size_t realSize = arraySize * arraySize;

	// Create an array of ${arraySize} integers
	unsigned int *arrayHostBase = new unsigned int[arraySize * arraySize];
	for (size_t i = 0; i < realSize; i++)
		arrayHostBase[i] = i;

	// initializing result for cpu
	unsigned int *resultCPU = new unsigned int[arraySize * arraySize];

	// calculating
	size_t resultIndex = 0;
	auto startCPU = std::chrono::steady_clock::now();

	// arraySize * 2 because we need to skip 2 rows
	for (size_t z = 0; z < arraySize * arraySize; z += arraySize * 2)
		// x+=2 because we need to skip 2 elements
		for (size_t x = 0; x < arraySize; x += 2) {
			resultCPU[resultIndex++] = arrayHostBase[z + x + arraySize + 1];
			resultCPU[resultIndex++] = arrayHostBase[z + x + arraySize];
			resultCPU[resultIndex++] = arrayHostBase[z + x + 1];
			resultCPU[resultIndex++] = arrayHostBase[z + x];
		}

	resultIndex = 0; // just in case
	auto stopCPU = std::chrono::steady_clock::now();
	auto millisecondsCPU = std::chrono::duration_cast<std::chrono::duration<float>>(stopCPU - startCPU).count() * 1000.0;
	cout << "CPU execution time: "
			 << millisecondsCPU
			 << " ms" << std::endl;

	// allocating memory on GPU
	unsigned int *resultGPU, *arrayGPU;
	hipMalloc((void **)&arrayGPU, realSize * sizeof(unsigned int));
	hipMalloc((void **)&resultGPU, realSize * sizeof(unsigned int));

	hipMemcpy(arrayGPU, arrayHostBase, realSize * sizeof(unsigned int), hipMemcpyHostToDevice); // copying data to GPU

	// calculating on GPU
	// calculating grid and block size
	unsigned int *gridBlockSize = calcGridBlockSize(arraySize);
	dim3 gridSize(gridBlockSize[0]);
	dim3 blockSize(gridBlockSize[1]);

	cout << "Blocks in grid: " << gridSize.x << std::endl
			 << "Threads in blocks: " << blockSize.x << std::endl;
	// Measure the time taken
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, nullptr);
	calculate<<<gridSize, blockSize>>>(arrayGPU, resultGPU, arraySize);
	hipEventRecord(stop, nullptr);
	hipEventSynchronize(stop);

	float millisecondsGPU = 0;
	hipEventElapsedTime(&millisecondsGPU, start, stop);

	std::cout << "Kernel execution time: " << millisecondsGPU << " ms" << std::endl;
	std::cout << "Ratio: " << millisecondsCPU / millisecondsGPU << std::endl;

	// copying data back to CPU
	unsigned int *bufferGPU = new unsigned int[realSize];
	hipMemcpy(bufferGPU, resultGPU, realSize * sizeof(unsigned int), hipMemcpyDeviceToHost);

	// report cuda state (error / no error)
	std::cerr << "Cuda status: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// output data into files
	if (!nfs) {
		//! \note This is a much faster way to write data into files. For 16*1024 elements it takes about 39.0~40.9~42.5 seconds to write data into files.
		//! \note Plus it adds newlines after every 4 elements.
		FILE *outputCPU = fopen("build/outputCPU.txt", "w");
		FILE *outputGPU = fopen("build/outputGPU.txt", "w");
		if (outputCPU && outputGPU) {
			auto startWriting = std::chrono::steady_clock::now();
			for (size_t i = 0; i < realSize; i += 4) {
				fprintf(outputCPU, "%u %u %u %u\n", *(resultCPU + i), *(resultCPU + i + 1), *(resultCPU + i + 2), *(resultCPU + i + 3));
				fprintf(outputGPU, "%u %u %u %u\n", *(bufferGPU + i), *(bufferGPU + i + 1), *(bufferGPU + i + 2), *(bufferGPU + i + 3));
			}
			auto stopWriting = std::chrono::steady_clock::now();
			cout << "Writing time: "
					 << std::chrono::duration_cast<std::chrono::duration<float>>(stopWriting - startWriting).count() * 1000.0
					 << " ms" << std::endl;
		} else
			cerr << "Failed to open one of output files!" << std::endl;
		if (outputCPU)
			fclose(outputCPU);
		if (outputGPU)
			fclose(outputGPU);
	}
	// free memory CPU
	delete[] arrayHostBase;
	delete[] resultCPU;
	delete[] bufferGPU;
	delete[] gridBlockSize;
	// free memory GPU
	hipFree(arrayGPU);
	hipFree(resultGPU);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
