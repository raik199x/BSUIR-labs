#include "hip/hip_runtime.h"
#include "../include/global.hpp"
#include "../include/thresholdTransform.h"
#include <cstring>

__global__ void thresholdPerform(short *d_image, short *result, size_t width, size_t height, short B, short G, short R, size_t threshold) {
	unsigned long index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index > width * height)
		return; // if index is out of range

	size_t resB = d_image[index * 3] - B;
	size_t resG = d_image[index * 3 + 1] - G;
	size_t resR = d_image[index * 3 + 2] - R;

	resB *= resB;
	resG *= resG;
	resR *= resR;

	size_t pixel = resB + resG + resR;
	// pixel        = sqrtf(pixel);

	// if (pixel > threshold) {
	if (pixel > (threshold * threshold)) {
		result[index * 3]     = 0;
		result[index * 3 + 1] = 0;
		result[index * 3 + 2] = 0;
	} else {
		result[index * 3]     = 255;
		result[index * 3 + 1] = 255;
		result[index * 3 + 2] = 255;
	}
}

short *performTransform(short *image, size_t width, size_t height, short B, short G, short R, size_t threshold, unsigned amountOfThreads) {
	// Allocating memory on the device
	short *d_image;
	hipHostMalloc((void **)&d_image, width * height * 3 * sizeof(short));
	//assert_v(hipMalloc((void **)&d_image, width * height * 3 * sizeof(short)) == hipSuccess, "hipMalloc failed (pt1)!");

	// Copying data from host to device
	memcpy(d_image, image, width * height * 3 * sizeof(short));
	//assert_v(hipMemcpy(d_image, image, width * height * 3 * sizeof(short), hipMemcpyHostToDevice) == hipSuccess, "hipMalloc failed (pt2)!");

	// Creating result buffer for GPU
	short *resultGPU;
	hipHostMalloc((void **)&resultGPU, width * height * 3 * sizeof(short));
	//assert_v(hipMalloc((void **)&resultGPU, width * height * 3 * sizeof(short)) == hipSuccess, "hipMalloc failed (pt3)!");

	// Calculating grid and block size
	dim3 gridSize(width * height / amountOfThreads + 1);
	dim3 blockSize(amountOfThreads);

	// Performing transform
	thresholdPerform<<<gridSize, blockSize>>>(d_image, resultGPU, width, height, B, G, R, threshold);

	// Check for cuda errors
	assert_v((std::string)hipGetErrorString(hipDeviceSynchronize()) == "no error", "Error: " + (std::string)hipGetErrorString(hipGetLastError()));

	// Copying data from device to host
	//assert_v(hipMemcpy(image, resultGPU, width * height * 3 * sizeof(short), hipMemcpyDeviceToHost) == hipSuccess, "hipMemcpy failed!");
	memcpy(image, resultGPU, width * height * 3 * sizeof(short));

	// Freeing memory
	//if (!expect_v(hipFree(d_image) == hipSuccess, "hipFree failed (A)!") || !expect_v(hipFree(resultGPU) == hipSuccess, "hipFree failed (B)!"))
	//	assert_v(false, "hipFree failed!");
	hipHostFree(d_image);
	hipHostFree(resultGPU);

	return image;
}
