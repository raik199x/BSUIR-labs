#include "hip/hip_runtime.h"
#include "../include/global.hpp"
#include "../include/houghMethod.h"

__global__ void HoughPerform(short *d_image, size_t *result, size_t width, size_t height, double ratio) {
	size_t index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index > width * height)
		return; // if index is out of range

	// skipping dark pixels
	if (d_image[index * 3] == 0 && d_image[index * 3 + 1] == 0 && d_image[index * 3 + 2] == 0)
		return;

	int x = 0, y = 0;
	// We have a problem, that we need to check whole line, but we don't know how long it is (since it's one dimensional array)
	// So we need to decided what is the actual row now (like is is was a 2D array) using a proportion
	// Current index  = x
	// Last index (width * height) = height
	// after that we round up the value to get current row
	size_t current_x_limit = (size_t)ceil((double)(index * height) / (width * height));
	// trying to calculate size on x axis
	for (size_t i = index; i < current_x_limit * width; i++)
		if (d_image[i * 3] == 255 && d_image[i * 3 + 1] == 255 && d_image[i * 3 + 2] == 255)
			x++;
		else
			break;

	if (x == 0)
		return;

	// trying to calculate size on y axis
	for (size_t i = index; i < width * height; i += width)
		if (d_image[i * 3] == 255 && d_image[i * 3 + 1] == 255 && d_image[i * 3 + 2] == 255)
			y++;
		else
			break;

	if (y == 0)
		return;

	double ratioGPU = (double)x / y;
	if (ratioGPU < ratio - 0.1 || ratioGPU > ratio + 0.1)
		return;

	x--;
	y--; //! For real dunno why it gets 1 pixel more
	// Now we need to check if there is 2 more lines that create rectangle
	// checking x axis (bottom left -> bottom right)
	for (size_t i = index + y * width; i < index + y * width + x; i++)
		if (d_image[i * 3] != 255 && d_image[i * 3 + 1] != 255 && d_image[i * 3 + 2] != 255)
			return;

	// checking y axis (top right -> bottom right)
	for (size_t i = index + x; i < index + x + y * width; i += width)
		if (d_image[i * 3] != 255 && d_image[i * 3 + 1] != 255 && d_image[i * 3 + 2] != 255)
			return;

	// if we are here, it means that we found 4 lines, so we can return
	if (result[0] != 0)
		return;
	result[0] = index;
	result[1] = x;
	result[2] = y;
}

size_t *performHough(short *image, size_t width, size_t height, unsigned amountOfThreads) {
	/*
	Due to the fact, that condition of the lab is very brief,
	we assume that we want to find exactly 4 lines, which are
	parallel to the x and y axis (and as a result the figure is a rectangle).
	*/

	// Allocating cuda memory
	short *d_image;
	assert_v(hipMalloc(&d_image, width * height * 3 * sizeof(short)) == hipSuccess, "hipMalloc failed!");

	// Copying data from host to device
	assert_v(hipMemcpy(d_image, image, width * height * 3 * sizeof(short), hipMemcpyHostToDevice) == hipSuccess, "hipMemcpy failed (host to device)!");

	// Creating result buffer for GPU
	size_t *resultGPU;
	assert_v(hipMalloc(&resultGPU, 3 * sizeof(size_t)) == hipSuccess, "hipMalloc failed!");
	assert_v(hipMemset(resultGPU, 0, 3 * sizeof(size_t)) == hipSuccess, "hipMemset failed (zeroes before kernel)!");

	// Calculating grid and block size
	dim3 gridSize(width * height / amountOfThreads + 1);
	dim3 blockSize(amountOfThreads);

	// Performing Haaf transform
	double ratio = 2.0 / 3.0; // as given in the lab (0.666667)
	HoughPerform<<<gridSize, blockSize>>>(d_image, resultGPU, width, height, ratio);
	assert_v((std::string)hipGetErrorString(hipDeviceSynchronize()) == "no error", "Haaf transform failed!");

	// Copying data from device to host
	size_t *result = new size_t[3];
	assert_v(hipMemcpy(result, resultGPU, 3 * sizeof(size_t), hipMemcpyDeviceToHost) == hipSuccess, "hipMemcpy failed (device to host)!");

	// Freeing memory
	if (!expect_v(hipFree(d_image) == hipSuccess, "hipFree failed (A)!") || !expect_v(hipFree(resultGPU) == hipSuccess, "hipFree failed (B)!"))
		assert_v(false, "hipFree failed!");

	if (result[0] == 0)
		return NULL; // if we didn't find anything, return NULL

	return result;
}
