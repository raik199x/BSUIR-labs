#include "hip/hip_runtime.h"
#include <filesystem>
#include <iostream>
#include <map>

#include <opencv2/imgcodecs.hpp>
#include <opencv2/opencv.hpp>
#include <utility>

#include "../include/cudaRotate.h"
#include "../include/global.hpp"
#include "../include/houghMethod.h"
#include "../include/opencvFunctions.h"
#include "../include/thresholdTransform.h"

// Condition 6
/*
marker = rectangular, color (RGB) C = (0 255 255)

(a) Используя пороговое преобразования обнаружить маркер цвета C согласно варианту.
(b) Используя преобразования Хафа определить параметры маркера;
(c) Используя выданное преобразование восстановить изображение.

*/
using namespace cv;

// since usually 1 pixel represents 8 bit, it's better to convert it to char (1 byte)
short *ConvertMatToShort(Mat inputImage) {
	// Get the image properties
	size_t rows      = inputImage.rows;
	size_t cols      = inputImage.cols;
	size_t channels  = inputImage.channels();
	size_t imageSize = rows * cols * channels;

	// Allocate memory on the CUDA device
	short *d_image = new short[imageSize];

	// copy the data from the OpenCV input image to the CUDA device memory
	for (size_t i = 0; i < imageSize; i++)
		d_image[i] = inputImage.data[i];

	return d_image;
}

Mat ConvertShortToMat(short *image, size_t width, size_t height, short channels) {
	size_t imageSize = width * height * channels;
	Mat result       = Mat(height, width, CV_8UC3);

	for (size_t i = 0; i < imageSize; i++)
		result.data[i] = image[i];
	return result;
}

void savingImage(std::string DirectoryNameForImages, std::string imageName, Mat image, std::string noLogs, std::string onlyResult, std::string LogMessage) {
	bool result;
	if (onlyResult == "true" && imageName == "/resultImage.jpeg")
		result = imwrite(DirectoryNameForImages + imageName, image);
	else if (onlyResult == "true")
		result = true;
	else if (onlyResult == "false")
		result = imwrite(DirectoryNameForImages + imageName, image);

	if (noLogs == "false") {
		if (result == true)
			std::cout << LogMessage << std::endl;
		else{
			std::cout << "Error with: " << LogMessage << std::endl;
			exit(1);
		}
	}
}

int main(int argc, char **argv) {
	// checkers
	if (argc < 2) {
		std::cout << "usage: ./run <Image path> <options>" << std::endl
		          << "Options:" << std::endl
		          << "\t-noLogs -- does not output anything in terminal" << std::endl
		          << "\t-onlyResult -- saves only result image" << std::endl
		          << "\t-savePath <path> -- saves images in given path" << std::endl
		          << "\t-threadsPerBlock <number> -- sets threads per block" << std::endl;
		return 0;
	}

	std::string filePath = argv[1];
	assert_v(std::filesystem::exists(filePath), "File does not exists");

	Mat image = imread(filePath);
	assert_v(image.total() <= pow(2, 32), "Image is too big");

	// inserting default options
	std::map<std::string, std::string> options;
	options.insert(std::make_pair("-noLogs", "false"));
	options.insert(std::make_pair("-onlyResult", "false"));
	options.insert(std::make_pair("-savePath", "imageProcessing"));
	options.insert(std::make_pair("-threadsPerBlock", "1024"));
	// parsing options
	for (int i = 2; i < argc; i++) {
		std::string option = argv[i];
		auto it            = options.find(option);
		if (it == options.end()) {
			std::cout << "Unknown option: " << option << std::endl;
			return 0;
		}
		if (option == "-threadsPerBlock") {
			i++;
			int threadsPerBlock;
			try {
				threadsPerBlock = std::stoul(argv[i]);
				if (threadsPerBlock > 1024)
					throw("Threads per block must be less than 1024");
				else if (threadsPerBlock == 0)
					throw("Threads per block must be greater than 0");
			} catch (std::string error) {
				std::cout << error << std::endl;
				return 0;
			}
			options[option] = std::to_string(threadsPerBlock);
		} else if (option == "-savePath")
			options[option] = argv[++i];
		else
			options[option] = "true";
	}

	// Configuration variables
	const std::string DirectoryNameForImages = options["-savePath"];
	// image is 2:3
	const size_t rectangleWidth  = 100 * 2;
	const size_t rectangleHeight = 100 * 3;
	// Since in lab given RGB format, and OpenCV get BGR format, we need to rotate
	const cv::Scalar rectangleColor(255, 255, 0);
	const unsigned threshold       = 100;
	const unsigned amountOfThreads = std::stoul(options["-threadsPerBlock"]);

	std::filesystem::create_directory(DirectoryNameForImages);
	// Saving original image
	savingImage(DirectoryNameForImages, "/0.originalImage.jpeg", image, options["-noLogs"], options["-onlyResult"], "Saving original Image");

	// Rotating image
	image = rotateImage(image, 55.0);
	savingImage(DirectoryNameForImages, "/1.rotatedImage.jpeg", image, options["-noLogs"], options["-onlyResult"], "OpenCV rotated image");
	Mat imageCopy = image.clone(); // saving for future use

	// Drawing rectangular
	image  = drawRectangular(image, rectangleColor, rectangleWidth, rectangleHeight);
	savingImage(DirectoryNameForImages, "/2.drawRectangular.jpeg", image, options["-noLogs"], options["-onlyResult"], "OpenCV: rectangular draw");

	// Drawing triangular
	image  = drawTriangular(image, rectangleColor, rectangleWidth, rectangleHeight);
	savingImage(DirectoryNameForImages, "/3.drawTriangular.jpeg", image, options["-noLogs"], options["-onlyResult"], "OpenCV: triangular draw");

	// threshold transformation
	Size size              = image.size();
	short *thresholdResult = performTransform(ConvertMatToShort(image), size.width, size.height, rectangleColor[0], rectangleColor[1], rectangleColor[2], threshold, amountOfThreads);

	// Create a cv::Mat object from the raw image data
	Mat thresholdResultMat = ConvertShortToMat(thresholdResult, size.width, size.height, 3);
	savingImage(DirectoryNameForImages, "/4.thresholdResult.jpeg", thresholdResultMat, options["-noLogs"], options["-onlyResult"], "CUDA: threshold transformation");

	// Hough transformation
	size_t *houghResult = performHough(thresholdResult, size.width, size.height, amountOfThreads);
	assert_v(houghResult != NULL, "Hough transformation failed");
	if(options["-noLogs"] == "false")
		std::cout << "CUDA: hough transformation successfully" << std::endl;

	// Creating image from hough transformation
	imageCopy = deleteArtifactsHough(imageCopy, houghResult, rectangleColor);
	savingImage(DirectoryNameForImages, "/5.HoughRestored.jpeg", imageCopy, options["-noLogs"], options["-onlyResult"], "CUDA + CPU + OpenCV: Image rectangle restored with hough");
	image = imageCopy.clone();

	// Rotating Image
	short *rotatedImage = performRotate(ConvertMatToShort(image), size.width, size.height, amountOfThreads);
	Mat rotateImageMat  = ConvertShortToMat(rotatedImage, size.width, size.height, 3);
	std::string resultImageName  = options["-onlyResult"] == "true" ? "/resultImage.jpeg" : "/6.rotateImage.jpeg";
	savingImage(DirectoryNameForImages, resultImageName, rotateImageMat, options["-noLogs"], options["-onlyResult"], "CUDA: image rotated successfully");

	return 0;
}
