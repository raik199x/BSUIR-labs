#include "hip/hip_runtime.h"
#include "../include/global.hpp"
#include "../include/cudaRotate.h"

__global__ void Rotate(short *Source, unsigned srcPitch, short *Destination, size_t width, size_t height, float angle) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid >= width * height)
		return; // if index is out of range

	int x = tid % width;
	int y = tid / width;

	float cosAngle = cos(angle * M_PI / 180.0f);
	float sinAngle = sin(angle * M_PI / 180.0f);

	int centerX = width / 2;
	int centerY = height / 2;

	int x1 = x - centerX;
	int y1 = y - centerY;

	int x2 = static_cast<int>(round(x1 * cosAngle - y1 * sinAngle));
	int y2 = static_cast<int>(round(x1 * sinAngle + y1 * cosAngle));

	int x3 = x2 + centerX;
	int y3 = y2 + centerY;

	if (x3 >= 0 && x3 < width && y3 >= 0 && y3 < height) {
		int tid2 = y3 * srcPitch + x3 * 3;

		Destination[tid * 3]     = Source[tid2];
		Destination[tid * 3 + 1] = Source[tid2 + 1];
		Destination[tid * 3 + 2] = Source[tid2 + 2];
	}
}

short *performRotate(short *image, size_t width, size_t height, unsigned amountOfThreads) {
	// Checking if width or height is 0
	assert_v(width != 0, "Width is 0");
	assert_v(height != 0, "Height is 0");

	// Allocating cuda memory
	short *d_image;
	size_t srcPitch;
	assert_v(hipMallocPitch(&d_image, &srcPitch, width * 3 * sizeof(short), height) == hipSuccess, "hipMalloc failed");

	// Copying data from host to device
	assert_v(hipMemcpy2D(d_image, srcPitch, image, width * 3 * sizeof(short), width * 3 * sizeof(short), height, hipMemcpyHostToDevice) == hipSuccess, "hipMemcpy failed (host to device)!");

	// Creating result buffer for GPU
	short *resultGPU;
	assert_v(hipMalloc(&resultGPU, height * width * 3 * sizeof(short)) == hipSuccess, "hipMalloc failed");

	// Calculating grid and block size
	dim3 gridSize(width * height / amountOfThreads + 1);
	dim3 blockSize(amountOfThreads);

	// Perform rotation
	float angle = 360 - 55;
	Rotate<<<gridSize, blockSize>>>(d_image, srcPitch / sizeof(short), resultGPU, width, height, angle);
	assert_v((std::string)hipGetErrorString(hipDeviceSynchronize()) == "no error", "Rotation failed!");

	// Copying data from device to host
	short *result = new short[width * 3 * height];
	assert_v(hipMemcpy(result, resultGPU, width * 3 * height * sizeof(short), hipMemcpyDeviceToHost) == hipSuccess, "hipMemcpy failed (device to host)!");

	// Freeing memory
	if (!expect_v(hipFree(d_image) == hipSuccess, "hipFree failed (A)") || !expect_v(hipFree(resultGPU) == hipSuccess, "hipFree failed (B)"))
		assert_v(false, "hipFree failed");

	return result;
}
