#include "hip/hip_runtime.h"
#include <opencv2/core/types.hpp>

#include "../include/opencvFunctions.h"

cv::Mat rotateImage(cv::Mat inputImage, double angle) {
	double radians = angle * CV_PI / 180.0;                           // converting to radians
	cv::Point2f center(inputImage.cols / 2.0, inputImage.rows / 2.0); // Get the center of rotation

	// Create the rotation matrix
	cv::Mat rotationMatrix = cv::getRotationMatrix2D(center, angle, 1.0);

	// Perform the rotation
	cv::Mat rotatedImage;
	cv::warpAffine(inputImage, rotatedImage, rotationMatrix, inputImage.size());

	return rotatedImage;
}

cv::Mat drawRectangular(cv::Mat inputImage, cv::Scalar rectangleColor, size_t rectangleWidth, size_t rectangleHeight) {
	// Get the center of the image
	cv::Point center(inputImage.cols / 2, inputImage.rows / 2);

	// Calculate the coordinates of the top-left and bottom-right corners of the rectangle
	cv::Point topLeft(center.x - rectangleWidth / 2, center.y - rectangleHeight / 2);
	cv::Point bottomRight(center.x + rectangleWidth / 2, center.y + rectangleHeight / 2);

	// Draw the rectangle on the image
	cv::rectangle(inputImage, topLeft, bottomRight, rectangleColor, 1, cv::LINE_AA);

	return inputImage;
}

cv::Mat drawTriangular(cv::Mat inputImage, cv::Scalar triangleColor, size_t triangleWidth, size_t triangleHeight) {
	// Get the height and width of the image
	int height = inputImage.rows;
	int width  = inputImage.cols;

	// Calculate the coordinates of the triangle vertices
	cv::Point pt1(inputImage.cols / 2, inputImage.rows / 2);
	cv::Point pt2(width - 2, height - 2); // Right bottom
	cv::Point pt3(2, height - 2);         // Left bottom

	// Draw the triangle by connecting the points with lines
	cv::line(inputImage, pt1, pt2, triangleColor, 1, cv::LINE_AA);
	cv::line(inputImage, pt2, pt3, triangleColor, 1, cv::LINE_AA);
	cv::line(inputImage, pt3, pt1, triangleColor, 1, cv::LINE_AA);

	return inputImage;
}

cv::Mat deleteArtifactsHough(cv::Mat inputImage, size_t *houghResult, cv::Scalar rectangleColor) {
	// Get the height and width of the image
	size_t height = inputImage.rows;
	size_t width  = inputImage.cols;

	// since houghResult[0] is taken from one dimensional array, we need to understand in which line it was taken
	// of course using proportional values
	size_t y = (size_t)ceil((double)(houghResult[0] * height) / (width * height));
	size_t x = (size_t)ceil((double)(houghResult[0] * width) / (width * height));

	// Calculate the coordinates of the triangle vertices
	cv::Point pt1(x, y);
	cv::Point pt2(x + houghResult[1], y);
	cv::Point pt3(x, y + houghResult[2]);
	cv::Point pt4(x + houghResult[1], y + houghResult[2]);

	// Draw the triangle by connecting the points with lines
	cv::line(inputImage, pt1, pt2, rectangleColor, 1, cv::LINE_AA);
	cv::line(inputImage, pt2, pt4, rectangleColor, 1, cv::LINE_AA);
	cv::line(inputImage, pt4, pt3, rectangleColor, 1, cv::LINE_AA);
	cv::line(inputImage, pt3, pt1, rectangleColor, 1, cv::LINE_AA);

	return inputImage;
}
