#include "hip/hip_runtime.h"
#include <cassert>

#include "../include/geometry.hpp"
#include "../include/volume.hpp"
#include "../include/cpu.hpp"
#include "../include/gpu.hpp"

/* cSpell:disable */
/*
• Точка O(0, 0, 0) всегда лежит внутри фигуры;
• Фигура всегда является выпуклым многогранником;
• В коде программы явно указаны входные точки и плоскости;
• При изменении координат точек программа должна вести себя корректно(при соблюдении предыдущих условий);
• Для вычислений использовать тип данных как минимум float. Для доп. задания можно использовать half;
• Разница в точности для CPU и GPU реализации не больше 0.001. Разница между ММК и аналитическим - 0.1;
• Для GPU должна использоваться разделяемая память;
• Для CPU измерение должно проводиться с помощью high − precision clock, для GPU - с помощью событий;
• Реализация на GPU должна использовать размер блока в пределах от 32 до максимально
допустимого вашей GPU. Программа должна поддерживать смену размера блока и по возможности вести себя валидно.
*/
/* cSpell:enable */
int main(const int argc, const char *const *argv) {
	bool noGpuInfo = true, defaultValues = true;
	string pointNames = "ABCDEF";
	if (argc == 1) {
		cout << "Lab 4: MMK cpu and gpu" << endl
		     << "Options: default - use default values (if set ignores setted points)" << endl
		     << "		  noGpuInfo - does not output info about GPU" << endl
		     << "		  <A,B,C,D,E,F> [<num>,<num>,<num>] - uses given values for mentioned point (if does not mentioned uses defaults)" << endl
		     << "examples: ./build/run A[-1, 0, 0.5] noGpuInfo" << endl;
			 return 0;
	}

	for (int i = 1; i < argc; ++i) {
		if (strcmp(argv[i], "default") == 0) {
			defaultValues = false;
		} else if (strcmp(argv[i], "noGpuInfo") == 0) {
			noGpuInfo = false;
		}
	}
	if (noGpuInfo)
		GPU::outputCudaInfo();
	// Working with points
	struct Geometry3D::Point points[6];
	// Setting default values (given by variant)
	points[0] = {-1, -1, -1};
	points[1] = {-1, 1, -1};
	points[2] = {1, -1, -1};
	points[3] = {1, 1, 1};
	points[4] = {-1, 1, 1};
	points[5] = {1, -1, 1};

	if (defaultValues)
		// Setting values from command line
		for (int i = 1; i < argc; ++i) {
			string line = argv[i];
			if (pointNames.find(line[0]) != string::npos) {
				int pointIndex       = pointNames.find(line[0]);
				line                 = line.substr(2, line.length() - 3);
				int commaIndex       = line.find(',');
				points[pointIndex].x = stof(line.substr(0, commaIndex));
				line                 = line.substr(commaIndex + 1, line.length() - commaIndex - 1);
				commaIndex           = line.find(',');
				points[pointIndex].y = stof(line.substr(0, commaIndex));
				line                 = line.substr(commaIndex + 1, line.length() - commaIndex - 1);
				points[pointIndex].z = stof(line);
			}
		}

	// Printing points
	cout << "Points:" << endl;
	for (int i = 0; i < 6; ++i) {
		cout << "\t" << pointNames[i] << ": " << points[i]() << endl;
	}
	// ended working with points

	//! \note Create hexahedron
	const Geometry3D::Polyhedron hexahedron = MonteCarlo::Hexahedron(points[0], points[1], points[2], points[3], points[4], points[5]);
	//! \note Extract vertices
	const std::vector<Geometry3D::Point> vertices = hexahedron.vertices();
	const size_t n_vertices = vertices.size();
	if (n_vertices != 8) {
		cerr << "Oops... Seems like number of vertices (" << n_vertices << ") doesn't equal to 8. Do better." << endl;
		return 1;
	}
	//! \note Find the most distant vertex
	const Geometry3D::Point most_distant_vertex = hexahedron.most_distant();
	//! \note Get the most distant from ORIGIN coordinate
	const xyz_type max_distance = std::max(abs(most_distant_vertex.x), std::max(abs(most_distant_vertex.y), abs(most_distant_vertex.z)));
	//! \note Get the PRNG limit by multiplying max_distance by MULTIPLIER_COEFFICIENT
	const xyz_type prng_limit = max_distance * MULTIPLIER_COEFFICIENT;
	const xyz_type cube_volume = pow((2 * prng_limit), 3);

	assert(hexahedron.planes.size() == 6);
	const Geometry3D::Plane planes[6] = {hexahedron.planes[0], hexahedron.planes[1], hexahedron.planes[2],
										 hexahedron.planes[3], hexahedron.planes[4], hexahedron.planes[5]};
	
	// doing tasks

	const xyz_type ratio_CPU = CPU::MonteCarlo(planes, prng_limit, N_OF_ITERATIONS);
	cout << "CPU ratio: " << ratio_CPU << endl;
	const xyz_type volume_CPU = cube_volume * ratio_CPU;
	cout << "CPU Volume: " << volume_CPU << endl;
	// not const since might be changed using cli (must be lower than 1024)
	size_t AmountOfThreads = 1024;
	const xyz_type ratio_GPU = GPU::initCudaCalc(planes, prng_limit, AmountOfThreads);
	cout << "GPU ratio: " << ratio_GPU << endl;
	const xyz_type volume_GPU = cube_volume * ratio_GPU;
	cout << "GPU Volume: " << volume_GPU << endl;

	return 0;
}