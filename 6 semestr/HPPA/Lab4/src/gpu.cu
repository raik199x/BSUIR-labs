#include "hip/hip_runtime.h"
#include "../include/gpu.hpp"
#include <array>

#include <cstddef>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void kernel(xyz_type prng_limit, size_t number_of_iterations, GPU::Plane *const gpuPlanes, unsigned long long *const device_result) {
	size_t currentIndex = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (currentIndex > number_of_iterations)
		return;
	//__shared__ size_t hit[1024];
	// hit[threadIdx.x] = 0;
	__syncthreads();

	hiprandState state;
	hiprand_init(clock64(), currentIndex, 0, &state);

	struct coordinates {
		xyz_type x;
		xyz_type y;
		xyz_type z;
	};
	struct coordinates rCoord;

	// the problem of hiprand is that it generates values from [0,1), so we need to manipulate values
	rCoord.x = prng_limit * hiprand_uniform(&state);
	rCoord.y = prng_limit * hiprand_uniform(&state);
	rCoord.z = prng_limit * hiprand_uniform(&state);

	float temp[3];
	temp[0] = hiprand_uniform(&state);
	temp[1] = hiprand_uniform(&state);
	temp[2] = hiprand_uniform(&state);

	if(temp[0] < 0.4)
		rCoord.x *= -1;
	if(temp[1] < 0.4)
		rCoord.y *= -1;
	if(temp[2] < 0.4)
		rCoord.z *= -1;

	// checking if point is in a polyhedron
	bool isInside = true;
	for (size_t i = 0; i < 6; ++i) {
		const xyz_type value = gpuPlanes[i].a * rCoord.x + gpuPlanes[i].b * rCoord.y + gpuPlanes[i].c * rCoord.z + gpuPlanes[i].d;
		if (value < 0) {
			isInside = false;
			break;
		}
	}
	if (isInside)
		atomicAdd(device_result, 1llu);
	
	// hit[threadIdx.x] = 1;

	//__syncthreads();
	//if (currentIndex == number_of_iterations - 1) {
	//	printf("block(%d, %d) thread(%d, %d): %f / %ld\n",
	//	       blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,
	//	       (*device_result), number_of_iterations);
	//	(*device_result) = (*device_result) / number_of_iterations;
	//}
}

std::array<size_t, 2> GPU::calculateDim(size_t AmountOfThreads) {
	std::array<size_t, 2> result;
	if (N_OF_ITERATIONS < AmountOfThreads) {
		result[0] = N_OF_ITERATIONS;
		result[1] = 1;
	} else {
		result[0] = AmountOfThreads;
		result[1] = (N_OF_ITERATIONS + AmountOfThreads - 1) / AmountOfThreads;
	}
	return result;
}

xyz_type GPU::initCudaCalc(const Geometry3D::Plane planes[6], const xyz_type prng_limit, size_t AmountOfThreads) {
	// Allocate memory for the result on the device
	unsigned long long *device_result;
	hipMalloc((void **)&device_result, sizeof(unsigned long long));
	hipMemset(device_result, 0, sizeof(float));

	// Since we cannot send structure with methods of host, we need to recreate struct that will contain only data
	Plane gpuPlanes[6];
	for (int i = 0; i < 6; i++)
		gpuPlanes[i] = {planes[i].a, planes[i].b, planes[i].c, planes[i].d};

	// Allocate memory for the struct on the device
	Plane *devPlanes;
	hipMalloc(&devPlanes, 6 * sizeof(Plane));
	// Copy the data from the host to the device
	hipMemcpy(devPlanes, gpuPlanes, 6 * sizeof(Plane), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	std::array<size_t, 2> size = calculateDim(AmountOfThreads);
	dim3 threads(size[0]);
	dim3 amountOfBlocks(size[1]);

	hipEventRecord(start, nullptr);
	kernel<<<amountOfBlocks, threads>>>(prng_limit, N_OF_ITERATIONS, devPlanes, device_result);
	hipDeviceSynchronize(); // ensure all CUDA work has completed

	hipEventRecord(stop, nullptr);
	hipEventSynchronize(stop);

	float millisecondsGPU = 0;
	hipEventElapsedTime(&millisecondsGPU, start, stop);
	std::cout << "Kernel execution time: " << millisecondsGPU << " ms" << std::endl;

	// report cuda state (error / no error)
	std::cerr << "Cuda status: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// Copy the result from the device to the host
	unsigned long long *host_result = new unsigned long long;
	hipMemcpy(host_result, device_result, sizeof(unsigned long long), hipMemcpyDeviceToHost);

	hipFree(device_result);
	hipFree(devPlanes);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (long double)(*host_result) / N_OF_ITERATIONS;
}

void GPU::outputCudaInfo(void) {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; ++i) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);
		cout << "\tDevice " << i << ": " << deviceProp.name << endl;
		cout << "\tTotal global memory: " << deviceProp.totalGlobalMem << endl;
		cout << "\tShared memory per block: " << deviceProp.sharedMemPerBlock << endl;
		cout << "\tMajor revision number: " << deviceProp.major << endl;
		cout << "\tMinor revision number: " << deviceProp.minor << endl;
	}
}
