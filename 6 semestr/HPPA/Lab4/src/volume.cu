#include "hip/hip_runtime.h"
#include <cmath>

#include "../include/time.hpp"
#include "../include/volume.hpp"
#include "../include/random.hpp"

using namespace Geometry3D;

/*!
 * \fn Tetrahedron(const Point& A, const Point& B, const Point& C, const Point& D) -> const Polyhedron
 * \brief Construct a tetrahedron from 4 points
 * \param A First point
 * \param B Second point
 * \param C Third point
 * \param D Fourth point
 * \return Tetrahedron
 */
const Polyhedron MonteCarlo::Tetrahedron(const Point& A, const Point& B, const Point& C, const Point& D) {
	//! \note construct planes from points
	Plane ABC = {A, B, C};
	Plane ABD = {A, B, D};
	Plane BCD = {B, C, D};
	Plane CAD = {C, A, D};
	//! \note construct polyhedron from planes
	Polyhedron tetrahedron = {ABC, ABD, BCD, CAD};
	tetrahedron.calibrate();
	return tetrahedron;
}

/*!
 * \fn Hexahedron(const Point& A, const Point& B, const Point& C, const Point& E, const Point& F, const Point& G) -> const Polyhedron
 * \brief Construct a hexahedron from 6 points
 * \param A First point
 * \param B Second point
 * \param C Third point
 * \param E Fourth point
 * \param F Fifth point
 * \param G Sixth point
 * \return Hexahedron
 */
const Polyhedron MonteCarlo::Hexahedron(const Point& A, const Point& B, const Point& C, const Point& E, const Point& F, const Point& G) {
	//! \note construct planes from points
	Plane ABC = {A, B, C};
	Plane ABF = {A, B, F};
	Plane ACG = {A, C, G};
	Plane FEG = {F, E, G};
	Plane EGC = {E, G, C};
	//! \note Should be EFD plane, but point D was not given
	//! \note in the task description so I used point B from
	//! \note the same plane instead. It's not a problem.
	Plane EFB = {E, F, B};
	//! \note construct polyhedron from planes
	Polyhedron hexahedron = {ABC, ABF, ACG, FEG, EGC, EFB};
	hexahedron.calibrate();
	return hexahedron;
}

const Polyhedron MonteCarlo::Pentahedron(const Point& A, const Point& B, const Point& C, const Point& D, const Point S) {
	//! \note construct planes from points
	Plane ABC = {A, B, C};
	Plane ABS = {A, B, S};
	Plane BCS = {B, C, S};
	Plane CDS = {C, D, S};
	Plane DAS = {D, A, S};
	//! \note construct polyhedron from planes
	Polyhedron pentahedron = {ABC, ABS, BCS, CDS, DAS};
	pentahedron.calibrate();
	return pentahedron;
}

/*!
 * \fn get_max_coordinate(const Point& point) -> xyz_type
 * \brief Returns maximum coordinate of the given point.
 * 
 * \param point Point to get maximum coordinate of.
 * \return Maximum coordinate of the given point.
 */
xyz_type get_max_coordinate(const Point& point) {
	return std::max(std::abs(point.x), std::max(std::abs(point.y), std::abs(point.z)));
}

/*!
 * \fn method(const Polyhedron& polyhedron) -> xyz_type
 * \brief Calculates volume of the given polyhedron using Monte Carlo method.
 * 
 * \param polyhedron Polyhedron to calculate volume of.
 * \return Volume of the given polyhedron.
 */
xyz_type MonteCarlo::method(const Polyhedron &polyhedron) {
	//! \note start clock
	const Clock start = Instant::now();

	//! \note Find the most distant vertex from the ORIGIN, aka (0, 0, 0)
	const auto most_distant_vertex = polyhedron.most_distant();
	cout << "\tMost distant vertex: " << most_distant_vertex() << endl;
	//! \note Calculate the limit of PRN generation (half of the cube's side)
	const auto limit = get_max_coordinate(most_distant_vertex);
	cout << "\tLimit: " << limit << endl;
	//! \note Multiply the limit by the coefficient to get the new limit (to avoid the case when the original volume equals the new one, happens when the polyhedron is a cube)
	const auto new_limit = limit * MULTIPLIER_COEFFICIENT;
	cout << "\tNew limit: " << new_limit << endl;
	//! \note Calculate the side of the new cube
	const auto cube_side = 2 * new_limit;
	//! \note Calculate the volume of the new cube
	const auto new_volume = cube_side * cube_side * cube_side;

	//! \note Start Monte Carlo method -> iterate N_OF_ITERATIONS times
	size_t counter = 0;
	for (size_t iteration = 0; iteration < N_OF_ITERATIONS; ++iteration) {
		//! \note Generate a random point in the new cube (random coordinates in the range [-new_limit, new_limit])
		const Point point =  {
			Random::Uniform::real<xyz_type>(-new_limit, new_limit),
			Random::Uniform::real<xyz_type>(-new_limit, new_limit),
			Random::Uniform::real<xyz_type>(-new_limit, new_limit)
		};
		//! \note Check if the generated point is inside the polyhedron
		if (polyhedron.has(point))
			//! \note Aka `Hit` -> the point is inside the polyhedron -> increment the counter
			++counter;
	}
	//! \note Calculate the volume of the polyhedron using the counter and the volume of the new cube
	const auto hit_ratio = (xyz_type)counter / N_OF_ITERATIONS;

	//! \note stop clock
	const Clock stop = Instant::now();
	//! \note print the time duration
	const Duration<std::milli> duration = stop - start;
	cout << "\tDuration: " << duration.count() << " ms" << endl;
	cout << "\tHit ratio: " << hit_ratio * 100.0l << "%" << endl;
	return new_volume * hit_ratio;
}

xyz_type Canonical::Tetrahedron(const Point& a, const Point& b, const Point& c, const Point& d) {
	//! \note Calculate the volume of the tetrahedron
	//! \note Find Vectors AD, AB, AC
	const Vector ad = {a, d};
	const Vector ab = {a, b};
	const Vector ac = {a, c};
	//! \note Calculate the volume of the tetrahedron
	const Vector cross = Vector::cross(ab, ac);
	const auto dot = Vector::dot(ad, cross);
	const auto volume = dot / 6;
	return volume < 0 ? -volume : volume;
}

xyz_type Canonical::Hexahedron(const std::vector<Point>& vertices) {
	//! \note Calculate the volume of the hexahedron
	const size_t n = vertices.size();
	xyz_type volume = 0;
	for (size_t i = 0; i < n; ++i)
		for (size_t j = 0; j < n; ++j)
			for (size_t k = 0; k < n; ++k)
				for (size_t l = 0; l < n; ++l) {
					if (vertices[i] == vertices[j] || vertices[i] == vertices[k] || vertices[i] == vertices[l] ||
						vertices[j] == vertices[i] || vertices[j] == vertices[k] || vertices[j] == vertices[l] ||
						vertices[k] == vertices[i] || vertices[k] == vertices[j] || vertices[k] == vertices[l] ||
						vertices[l] == vertices[i] || vertices[l] == vertices[j] || vertices[l] == vertices[k])
						continue;
					if (Plane(vertices[j], vertices[k], vertices[l]).has(vertices[i]) ||
						Plane(vertices[i], vertices[k], vertices[l]).has(vertices[j]) ||
						Plane(vertices[i], vertices[j], vertices[l]).has(vertices[k]) ||
						Plane(vertices[i], vertices[j], vertices[k]).has(vertices[l]))
						continue;
					volume += Tetrahedron(vertices[i], vertices[j], vertices[k], vertices[l]);
				}
	return volume / 240;
}

xyz_type Canonical::Pentahedron(const std::vector<Point>& vertices) {
	//! \note Calculate the volume of the pyramid
	const size_t n = vertices.size();
	xyz_type volume = 0;
	for (size_t i = 0; i < n; ++i)
		for (size_t j = 0; j < n; ++j)
			for (size_t k = 0; k < n; ++k)
				for (size_t l = 0; l < n; ++l) {
					if (vertices[i] == vertices[j] || vertices[i] == vertices[k] || vertices[i] == vertices[l] ||
						vertices[j] == vertices[i] || vertices[j] == vertices[k] || vertices[j] == vertices[l] ||
						vertices[k] == vertices[i] || vertices[k] == vertices[j] || vertices[k] == vertices[l] ||
						vertices[l] == vertices[i] || vertices[l] == vertices[j] || vertices[l] == vertices[k])
						continue;
					if (Plane(vertices[j], vertices[k], vertices[l]).has(vertices[i]) ||
						Plane(vertices[i], vertices[k], vertices[l]).has(vertices[j]) ||
						Plane(vertices[i], vertices[j], vertices[l]).has(vertices[k]) ||
						Plane(vertices[i], vertices[j], vertices[k]).has(vertices[l]))
						continue;
					volume += Tetrahedron(vertices[i], vertices[j], vertices[k], vertices[l]);
				}
	return volume / 48;
}
