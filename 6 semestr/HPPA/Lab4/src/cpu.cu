#include "hip/hip_runtime.h"
#include "../include/cpu.hpp"
#include "../include/time.hpp"
#include "../include/random.hpp"
#include <ratio>

using namespace Geometry3D;

xyz_type prng(const xyz_type min, const xyz_type max) {
	return Random::Uniform::real<xyz_type>(min, max);
}

bool polyhedron_has_point(const Plane planes[6], const Point& p) {
	for (size_t i = 0; i < 6; ++ i) {
		const xyz_type value = planes[i].a * p.x + planes[i].b * p.y + planes[i].c * p.z + planes[i].d;
		if (value < 0)
			//! \note Does not belong to the Polyhedron
			return false;
	}
	//! \note Does belong to the Polyhedron
	return true;
}

xyz_type CPU::MonteCarlo(const Plane planes[6], const xyz_type prng_limit, const size_t number_of_iterations) {
	//! \note start clock
	const Clock start = Instant::now();

	//! \note Start Monte Carlo method -> iterate N_OF_ITERATIONS times
	size_t counter = 0;
	for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
		//! \note Generate a random point in the new cube (random coordinates in the range [-prng_limit, prng_limit])
		const Point point = {
			prng(-prng_limit, prng_limit),
			prng(-prng_limit, prng_limit),
			prng(-prng_limit, prng_limit)
		};
		//! \note Check if the generated point is inside the polyhedron
		if (polyhedron_has_point(planes, point))
			//! \note Aka `Hit` -> the point is inside the polyhedron -> increment the counter
			++counter;
	}
	//! \note Calculate the hit ratio
	const auto ratio = (xyz_type)counter / number_of_iterations;

	//! \note stop clock
	const Clock stop = Instant::now();
	//! \note calculate time duration
	const Duration<std::milli> duration = stop - start;
	cout << "CPU time: " << duration.count() << "ms" << endl;
	return ratio;
}
